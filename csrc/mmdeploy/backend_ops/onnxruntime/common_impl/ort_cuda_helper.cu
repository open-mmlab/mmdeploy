#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.
#include "common_cuda_helper.hpp"
#include <onnxruntime_cxx_api.h>
#include <hip/hip_fp16.h>

namespace mmdeploy {
  const int MAXTENSORDIMS = 10;

  struct TensorDesc {
    int shape[MAXTENSORDIMS];
    int stride[MAXTENSORDIMS];
    int dim;
  };
}

using mmdeploy::TensorDesc;

template <class scalar_t>
__global__ void copy_permute_kernel(scalar_t *__restrict__ dst, const scalar_t *__restrict__ src,
                                    int n, TensorDesc ts_src_stride, TensorDesc ts_dst_stride,
                                    TensorDesc ts_permute) {
  const int src_dim = ts_src_stride.dim;
  const auto src_stride = ts_src_stride.stride;
  const auto dst_stride = ts_dst_stride.stride;
  const auto permute = ts_permute.shape;
  CUDA_1D_KERNEL_LOOP(index, n) {
    size_t dst_index = index;
    size_t src_index = 0;
    for (int i = 0; i < src_dim; ++i) {
      int dim_index = dst_index / dst_stride[i];
      dst_index = dst_index % dst_stride[i];
      src_index += dim_index * src_stride[permute[i]];
    }
    dst[index] = src[src_index];
  }
}

template <class scalar_t>
void memcpyPermute(scalar_t *dst, const scalar_t *src, int *src_size, int *permute, int src_dim,
                   hipStream_t stream) {
  size_t copy_size = 1;
  TensorDesc ts_permute;
  memcpy(&(ts_permute.shape[0]), permute, src_dim * sizeof(int));

  TensorDesc ts_src_stride;
  TensorDesc ts_dst_stride;
  ts_src_stride.dim = src_dim;
  ts_dst_stride.dim = src_dim;
  int *src_stride = &(ts_src_stride.stride[0]);
  int *dst_stride = &(ts_dst_stride.stride[0]);
  int *dst_size = &(ts_dst_stride.shape[0]);
  src_stride[src_dim - 1] = 1;
  dst_stride[src_dim - 1] = 1;

  for (int i = src_dim - 1; i >= 0; --i) {
    dst_size[i] = src_size[permute[i]];
    if (i < src_dim - 1) {
      src_stride[i] = src_stride[i + 1] * src_size[i + 1];
    }
  }

  for (int i = src_dim - 1; i >= 0; --i) {
    copy_size *= dst_size[i];
    if (i < src_dim - 1) {
      dst_stride[i] = dst_stride[i + 1] * dst_size[i + 1];
    }
  }

  copy_permute_kernel<scalar_t><<<GET_BLOCKS(copy_size), THREADS_PER_BLOCK, 0, stream>>>(
      dst, src, copy_size, ts_src_stride, ts_dst_stride, ts_permute);
}

template void memcpyPermute<float>(float *dst, const float *src, int *src_size, int *permute,
                                   int src_dim, hipStream_t stream);
template void memcpyPermute<__half>(__half *dst, const __half *src, int *src_size, int *permute,
                                  int src_dim, hipStream_t stream);


template <>
hipblasStatus_t cublasGemmWrap<float>(hipblasHandle_t handle, hipblasOperation_t transa,
                                     hipblasOperation_t transb, int m, int n, int k,
                                     const float *alpha, const float *A, int lda, const float *B,
                                     int ldb, const float *beta, float *C, int ldc) {
  return hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

template <>
hipblasStatus_t cublasGemmWrap<__half>(hipblasHandle_t handle, hipblasOperation_t transa,
                                    hipblasOperation_t transb, int m, int n, int k,
                                    const __half *alpha, const __half *A, int lda, const __half *B,
                                    int ldb, const __half *beta, __half *C, int ldc) {
  return hipblasHgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
