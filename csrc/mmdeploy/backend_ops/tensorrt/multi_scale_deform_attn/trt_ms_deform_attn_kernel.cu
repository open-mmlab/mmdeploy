#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved
#include <assert.h>
#include <hip/hip_fp16.h>

#include "common_cuda_helper.hpp"
#include "trt_ms_deform_attn_kernel.cuh"
#include "trt_ms_deform_attn_kernel.hpp"
#include "trt_plugin_helper.hpp"

template<typename scalar_t>
void ms_deformable_im2col_cuda(hipStream_t    stream,
                               scalar_t const* dataValue,
                               int32_t const*  dataSpatialShapes,
                               int32_t const*  dataLevelStartIndex,
                               scalar_t const* dataSamplingLoc,
                               scalar_t const* dataAttnWeight,
                               int32_t const   batchSize,
                               int32_t const   spatialSize,
                               int32_t const   numHeads,
                               int32_t const   channels,
                               int32_t const   numLevels,
                               int32_t const   numQuery,
                               int32_t const   numPoint,
                               scalar_t*       dataCol)
{
    int32_t const numKernels       = batchSize * numQuery * numHeads * channels;
    int32_t const numActualKernels = batchSize * numQuery * numHeads * channels;

    ms_deformable_im2col_gpu_kernel<scalar_t>
        <<<GET_BLOCKS(numActualKernels), THREADS_PER_BLOCK, 0, stream>>>(numKernels,
                                                                         dataValue,
                                                                         dataSpatialShapes,
                                                                         dataLevelStartIndex,
                                                                         dataSamplingLoc,
                                                                         dataAttnWeight,
                                                                         batchSize,
                                                                         spatialSize,
                                                                         numHeads,
                                                                         channels,
                                                                         numLevels,
                                                                         numQuery,
                                                                         numPoint,
                                                                         dataCol);
}

template<typename scalar_t>
int32_t ms_deform_attn_cuda_forward(const scalar_t* value,
                                    const int32_t*  spatialShapes,
                                    const int32_t*  levelStartIndex,
                                    const scalar_t* samplingLoc,
                                    const scalar_t* attnWeight,
                                    scalar_t*       output,
                                    int32_t         batch,
                                    int32_t         mSpatialSize,
                                    int32_t         mNumHeads,
                                    int32_t         mChannels,
                                    int32_t         mNumLevels,
                                    int32_t         mNumQuery,
                                    int32_t         mNumPoint,
                                    hipStream_t    stream)
{
    auto    perValueSize      = mSpatialSize * mNumHeads * mChannels;
    auto    perSampleLocSize  = mNumQuery * mNumHeads * mNumLevels * mNumPoint * 2;
    auto    perAttnWeightSize = mNumQuery * mNumHeads * mNumLevels * mNumPoint;
    auto    perOutputSize     = mNumQuery * mNumHeads * mChannels;

    int32_t mIm2colStep = batch;

    for (int32_t n = 0; n < batch / mIm2colStep; ++n)
    {
        auto columns = output + n * mIm2colStep * perOutputSize;
        ms_deformable_im2col_cuda<scalar_t>(stream,
                                            value + n * mIm2colStep * perValueSize,
                                            spatialShapes,
                                            levelStartIndex,
                                            samplingLoc + n * mIm2colStep * perSampleLocSize,
                                            attnWeight + n * mIm2colStep * perAttnWeightSize,
                                            mIm2colStep,
                                            mSpatialSize,
                                            mNumHeads,
                                            mChannels,
                                            mNumLevels,
                                            mNumQuery,
                                            mNumPoint,
                                            columns);
    }

    return 0;
}

template int32_t ms_deform_attn_cuda_forward<float>(const float*   value,
                                                    const int32_t* spatialShapes,
                                                    const int32_t* levelStartIndex,
                                                    const float*   samplingLoc,
                                                    const float*   attnWeight,
                                                    float*         output,
                                                    int32_t        batch,
                                                    int32_t        mSpatialSize,
                                                    int32_t        mNumHeads,
                                                    int32_t        mChannels,
                                                    int32_t        mNumLevels,
                                                    int32_t        mNumQuery,
                                                    int32_t        mNumPoint,
                                                    hipStream_t   stream);

template int32_t ms_deform_attn_cuda_forward<__half>(const __half*  value,
                                                     const int32_t* spatialShapes,
                                                     const int32_t* levelStartIndex,
                                                     const __half*  samplingLoc,
                                                     const __half*  attnWeight,
                                                     __half*        output,
                                                     int32_t        batch,
                                                     int32_t        mSpatialSize,
                                                     int32_t        mNumHeads,
                                                     int32_t        mChannels,
                                                     int32_t        mNumLevels,
                                                     int32_t        mNumQuery,
                                                     int32_t        mNumPoint,
                                                     hipStream_t   stream);
