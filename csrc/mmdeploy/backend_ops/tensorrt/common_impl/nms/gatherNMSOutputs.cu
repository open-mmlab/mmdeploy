#include "hip/hip_runtime.h"
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
// modify from
// https://github.com/NVIDIA/TensorRT/tree/master/plugin/batchedNMSPlugin
#include <vector>

#include "nms/kernel.h"
#include "trt_plugin_helper.hpp"

template <typename T_BBOX, typename T_SCORE, bool rotated, unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta) __global__
    void gatherNMSOutputs_kernel(const bool shareLocation, const int numImages,
                                 const int numPredsPerClass, const int numClasses, const int topK,
                                 const int keepTopK, const int *indices, const T_SCORE *scores,
                                 const T_BBOX *bboxData, T_BBOX *nmsedDets, int *nmsedLabels,
                                 int *nmsedIndex, bool clipBoxes) {
  if (keepTopK > topK) return;
  for (int i = blockIdx.x * nthds_per_cta + threadIdx.x; i < numImages * keepTopK;
       i += gridDim.x * nthds_per_cta) {
    const int imgId = i / keepTopK;
    const int detId = i % keepTopK;
    const int offset = imgId * numClasses * topK;
    const int index = indices[offset + detId];
    const T_SCORE score = scores[offset + detId];
    if (index == -1) {
      nmsedLabels[i] = -1;
      if (nmsedIndex != nullptr) {
        nmsedIndex[i] = -1;
      }
      if (rotated) {
        nmsedDets[i * 6] = 0;
        nmsedDets[i * 6 + 1] = 0;
        nmsedDets[i * 6 + 2] = 0;
        nmsedDets[i * 6 + 3] = 0;
        nmsedDets[i * 6 + 4] = 0;
        nmsedDets[i * 6 + 5] = 0;
      } else {
        nmsedDets[i * 5] = 0;
        nmsedDets[i * 5 + 1] = 0;
        nmsedDets[i * 5 + 2] = 0;
        nmsedDets[i * 5 + 3] = 0;
        nmsedDets[i * 5 + 4] = 0;
      }
    } else {
      const int bboxOffset =
          imgId * (shareLocation ? numPredsPerClass : (numClasses * numPredsPerClass));
      nmsedLabels[i] = (index % (numClasses * numPredsPerClass)) / numPredsPerClass;  // label
      if (rotated) {
        const int bboxId = ((shareLocation ? (index % numPredsPerClass)
                                           : index % (numClasses * numPredsPerClass)) +
                            bboxOffset) *
                           5;
        if (nmsedIndex != nullptr) {
          nmsedIndex[i] = bboxId / 5;
        }
        // clipped bbox xmin
        nmsedDets[i * 6] =
            clipBoxes ? max(min(bboxData[bboxId], T_BBOX(1.)), T_BBOX(0.)) : bboxData[bboxId];
        // clipped bbox ymin
        nmsedDets[i * 6 + 1] = clipBoxes ? max(min(bboxData[bboxId + 1], T_BBOX(1.)), T_BBOX(0.))
                                         : bboxData[bboxId + 1];
        // clipped bbox xmax
        nmsedDets[i * 6 + 2] = clipBoxes ? max(min(bboxData[bboxId + 2], T_BBOX(1.)), T_BBOX(0.))
                                         : bboxData[bboxId + 2];
        // clipped bbox ymax
        nmsedDets[i * 6 + 3] = clipBoxes ? max(min(bboxData[bboxId + 3], T_BBOX(1.)), T_BBOX(0.))
                                         : bboxData[bboxId + 3];
        // clipped bbox angle
        nmsedDets[i * 6 + 4] = clipBoxes ? max(min(bboxData[bboxId + 4], T_BBOX(1.)), T_BBOX(0.))
                                         : bboxData[bboxId + 4];
        nmsedDets[i * 6 + 5] = score;
      } else {
        const int bboxId = ((shareLocation ? (index % numPredsPerClass)
                                           : index % (numClasses * numPredsPerClass)) +
                            bboxOffset) *
                           4;
        if (nmsedIndex != nullptr) {
          nmsedIndex[i] = bboxId / 4;
        }
        // clipped bbox xmin
        nmsedDets[i * 5] =
            clipBoxes ? max(min(bboxData[bboxId], T_BBOX(1.)), T_BBOX(0.)) : bboxData[bboxId];
        // clipped bbox ymin
        nmsedDets[i * 5 + 1] = clipBoxes ? max(min(bboxData[bboxId + 1], T_BBOX(1.)), T_BBOX(0.))
                                         : bboxData[bboxId + 1];
        // clipped bbox xmax
        nmsedDets[i * 5 + 2] = clipBoxes ? max(min(bboxData[bboxId + 2], T_BBOX(1.)), T_BBOX(0.))
                                         : bboxData[bboxId + 2];
        // clipped bbox ymax
        nmsedDets[i * 5 + 3] = clipBoxes ? max(min(bboxData[bboxId + 3], T_BBOX(1.)), T_BBOX(0.))
                                         : bboxData[bboxId + 3];
        nmsedDets[i * 5 + 4] = score;
      }
    }
  }
}

template <typename T_BBOX, typename T_SCORE, bool rotated>
pluginStatus_t gatherNMSOutputs_gpu(hipStream_t stream, const bool shareLocation,
                                    const int numImages, const int numPredsPerClass,
                                    const int numClasses, const int topK, const int keepTopK,
                                    const void *indices, const void *scores, const void *bboxData,
                                    void *nmsedDets, void *nmsedLabels, void *nmsedIndex,
                                    bool clipBoxes) {
  const int BS = 32;
  const int GS = 32;
  gatherNMSOutputs_kernel<T_BBOX, T_SCORE, rotated, BS><<<GS, BS, 0, stream>>>(
      shareLocation, numImages, numPredsPerClass, numClasses, topK, keepTopK, (int *)indices,
      (T_SCORE *)scores, (T_BBOX *)bboxData, (T_BBOX *)nmsedDets, (int *)nmsedLabels,
      (int *)nmsedIndex, clipBoxes);

  CSC(hipGetLastError(), STATUS_FAILURE);
  return STATUS_SUCCESS;
}

// gatherNMSOutputs LAUNCH CONFIG {{{
typedef pluginStatus_t (*nmsOutFunc)(hipStream_t, const bool, const int, const int, const int,
                                     const int, const int, const void *, const void *, const void *,
                                     void *, void *, void *, bool);
struct nmsOutLaunchConfig {
  DataType t_bbox;
  DataType t_score;
  bool rotated;
  nmsOutFunc function;

  nmsOutLaunchConfig(DataType t_bbox, DataType t_score, bool rotated)
      : t_bbox(t_bbox), t_score(t_score), rotated(rotated) {}
  nmsOutLaunchConfig(DataType t_bbox, DataType t_score, bool rotated, nmsOutFunc function)
      : t_bbox(t_bbox), t_score(t_score), rotated(rotated), function(function) {}
  bool operator==(const nmsOutLaunchConfig &other) {
    return t_bbox == other.t_bbox && t_score == other.t_score && rotated == other.rotated;
  }
};

using nvinfer1::DataType;

static std::vector<nmsOutLaunchConfig> nmsOutFuncVec;

bool nmsOutputInit() {
  nmsOutFuncVec.push_back(nmsOutLaunchConfig(DataType::kFLOAT, DataType::kFLOAT, false,
                                             gatherNMSOutputs_gpu<float, float, false>));
  nmsOutFuncVec.push_back(nmsOutLaunchConfig(DataType::kFLOAT, DataType::kFLOAT, true,
                                             gatherNMSOutputs_gpu<float, float, true>));
  return true;
}

static bool initialized = nmsOutputInit();

pluginStatus_t gatherNMSOutputs(hipStream_t stream, const bool shareLocation, const int numImages,
                                const int numPredsPerClass, const int numClasses, const int topK,
                                const int keepTopK, const DataType DT_BBOX, const DataType DT_SCORE,
                                const void *indices, const void *scores, const void *bboxData,
                                void *nmsedDets, void *nmsedLabels, void *nmsedIndex,
                                bool clipBoxes, bool rotated) {
  nmsOutLaunchConfig lc = nmsOutLaunchConfig(DT_BBOX, DT_SCORE, rotated);
  for (unsigned i = 0; i < nmsOutFuncVec.size(); ++i) {
    if (lc == nmsOutFuncVec[i]) {
      DEBUG_PRINTF("gatherNMSOutputs kernel %d\n", i);
      return nmsOutFuncVec[i].function(stream, shareLocation, numImages, numPredsPerClass,
                                       numClasses, topK, keepTopK, indices, scores, bboxData,
                                       nmsedDets, nmsedLabels, nmsedIndex, clipBoxes);
    }
  }
  return STATUS_BAD_PARAM;
}
