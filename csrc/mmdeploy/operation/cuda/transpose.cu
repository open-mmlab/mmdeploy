// Copyright (c) OpenMMLab. All rights reserved.


#include <hip/hip_runtime.h>
#include <cstdint>

namespace mmdeploy {
namespace operation {
namespace cuda {

template <typename T>
__global__ void transpose(const T* src, int height, int width, int channels, int src_width_stride,
                          T* dst, int dst_channel_stride) {
  auto x = blockIdx.x * blockDim.x + threadIdx.x;
  auto y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height) return;

  for (auto c = 0; c < channels; ++c) {
    dst[c * dst_channel_stride + y * width + x] = src[y * src_width_stride + x * channels + c];
  }
}

template <typename T>
void Transpose(const T* src, int height, int width, int channels, T* dst, hipStream_t stream) {
  const dim3 thread_block(32, 8);
  const dim3 block_num((width + thread_block.x - 1) / thread_block.x,
                       (height + thread_block.y - 1) / thread_block.y);

  auto src_width_stride = width * channels;
  auto dst_channel_stride = width * height;

  transpose<T><<<block_num, thread_block, 0, stream>>>(src, height, width, channels,
                                                       src_width_stride, dst, dst_channel_stride);
}

template void Transpose<uint8_t>(const uint8_t* src, int height, int width, int channels,
                                 uint8_t* dst, hipStream_t stream);

template void Transpose<float>(const float* src, int height, int width, int channels, float* dst,
                               hipStream_t stream);
}  // namespace cuda
}  // namespace operation
}  // namespace mmdeploy
