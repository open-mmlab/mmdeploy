// Copyright (c) OpenMMLab. All rights reserved.

#include <hip/hip_runtime.h>

#include <cstdint>

namespace mmdeploy {
namespace operation {
namespace cuda {
namespace impl {

template <typename T, int channels>
__global__ void normalize(const T* src, int height, int width, int stride, float* output,
                          const float3 mean, const float3 std, bool to_rgb) {
  int x = (int)(blockIdx.x * blockDim.x + threadIdx.x);
  int y = (int)(blockIdx.y * blockDim.y + threadIdx.y);

  if (x >= width || y >= height) {
    return;
  }

  int loc = y * stride + x * channels;
  auto mean_ptr = &mean.x;
  auto std_ptr = &std.x;
  if (to_rgb) {
    for (int c = 0; c < channels; ++c) {
      output[loc + c] = ((float)src[loc + channels - 1 - c] - mean_ptr[c]) * std_ptr[c];
    }
  } else {
    for (int c = 0; c < channels; ++c) {
      output[loc + c] = ((float)src[loc + c] - mean_ptr[c]) * std_ptr[c];
    }
  }
}

template <typename T, int channels>
void Normalize(const T* src, int height, int width, int stride, float* output, const float* mean,
               const float* std, bool to_rgb, hipStream_t stream) {
  const dim3 thread_block(16, 16);
  const dim3 num_blocks((width + thread_block.x - 1) / thread_block.x,
                        (height + thread_block.y - 1) / thread_block.y);
  const float3 _mean{mean[0], mean[1], mean[2]};
  const float3 _std{float(1. / std[0]), float(1. / std[1]), float(1. / std[2])};
  normalize<T, channels><<<num_blocks, thread_block, 0, stream>>>(src, height, width, stride,
                                                                  output, _mean, _std, to_rgb);
}

template void Normalize<uint8_t, 3>(const uint8_t* src, int height, int width, int stride,
                                    float* output, const float* mean, const float* std, bool to_rgb,
                                    hipStream_t stream);
template void Normalize<uint8_t, 1>(const uint8_t* src, int height, int width, int stride,
                                    float* output, const float* mean, const float* std, bool to_rgb,
                                    hipStream_t stream);

template void Normalize<float, 3>(const float* src, int height, int width, int stride,
                                  float* output, const float* mean, const float* std, bool to_rgb,
                                  hipStream_t stream);
template void Normalize<float, 1>(const float* src, int height, int width, int stride,
                                  float* output, const float* mean, const float* std, bool to_rgb,
                                  hipStream_t stream);
}  // namespace impl
}  // namespace cuda
}  // namespace operation
}  // namespace mmdeploy
