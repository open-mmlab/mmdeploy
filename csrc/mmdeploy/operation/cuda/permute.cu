// Copyright (c) OpenMMLab. All rights reserved.


#include <hip/hip_runtime.h>
#include <cstdint>

namespace mmdeploy {
namespace operation {
namespace cuda {
namespace impl {

template <typename T>
__global__ void permute(const T* src, const int* src_strides, T* dst, const int* dst_strides,
                        int ndim, int total) {
  int u = blockIdx.x * blockDim.x + threadIdx.x;
  if (u >= total) {
    return;
  }

  int remaining = u;
  int v = 0;
  for (int i = 0; i < ndim; i++) {
    int p = remaining / dst_strides[i];
    remaining -= p * dst_strides[i];
    v += p * src_strides[i];
  }
  dst[u] = src[v];
}

template <typename T>
void Permute(const T* src, const int* src_strides, T* dst, const int* dst_strides, int ndim,
             int total, hipStream_t stream) {
  int thread_num = 256;
  int block_num = (total + thread_num - 1) / thread_num;
  permute<T><<<block_num, thread_num, 0, stream>>>(src, src_strides, dst, dst_strides, ndim, total);
}

template void Permute<float>(const float* src, const int* src_strides, float* dst,
                             const int* dst_strides, int ndim, int total, hipStream_t stream);

template void Permute<uint8_t>(const uint8_t* src, const int* src_strides, uint8_t* dst,
                               const int* dst_strides, int ndim, int total, hipStream_t stream);

}  // namespace impl
}  // namespace cuda
}  // namespace operation
}  // namespace mmdeploy
