// Copyright (c) OpenMMLab. All rights reserved.


#include <hip/hip_runtime.h>
#include <cstdint>

namespace mmdeploy {
namespace operation {
namespace cuda {
namespace impl {

template <typename T, int channels>
__global__ void crop(const T *src, int src_w, T *dst, int dst_h, int dst_w, int offset_h,
                     int offset_w) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dst_w || y >= dst_h) return;
  int src_x = x + offset_w;
  int src_y = y + offset_h;

  int dst_loc = (y * dst_w + x) * channels;
  int src_loc = (src_y * src_w + src_x) * channels;

  for (int i = 0; i < channels; ++i) {
    dst[dst_loc + i] = src[src_loc + i];
  }
}

template <typename T, int channels>
void Crop(const T *src, int src_w, T *dst, int dst_h, int dst_w, int offset_h, int offset_w,
          hipStream_t stream) {
  const dim3 thread_block(32, 8);
  const dim3 block_num((dst_w + thread_block.x - 1) / thread_block.x,
                       (dst_h + thread_block.y - 1) / thread_block.y);
  crop<T, channels>
      <<<block_num, thread_block, 0, stream>>>(src, src_w, dst, dst_h, dst_w, offset_h, offset_w);
}

template void Crop<uint8_t, 3>(const uint8_t *src, int src_w, uint8_t *dst, int dst_h, int dst_w,
                               int offset_h, int offset_w, hipStream_t stream);

template void Crop<uint8_t, 1>(const uint8_t *src, int src_w, uint8_t *dst, int dst_h, int dst_w,
                               int offset_h, int offset_w, hipStream_t stream);

template void Crop<float, 3>(const float *src, int src_w, float *dst, int dst_h, int dst_w,
                             int offset_h, int offset_w, hipStream_t stream);

template void Crop<float, 1>(const float *src, int src_w, float *dst, int dst_h, int dst_w,
                             int offset_h, int offset_w, hipStream_t stream);

}  // namespace impl
}  // namespace cuda
}  // namespace operation
}  // namespace mmdeploy
