#include "hip/hip_runtime.h"
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
// modify from
// https://github.com/NVIDIA/TensorRT/tree/master/plugin/batchedNMSPlugin
#include <stdint.h>

#include <hipcub/hipcub.hpp>

#include "hipblas.h"
#include "nms/kernel.h"
#include "trt_plugin_helper.hpp"

#define CUDA_MEM_ALIGN 256

// return cuda arch
size_t get_cuda_arch(int devID) {
  int computeMode = -1, major = 0, minor = 0;
  CUASSERT(hipDeviceGetAttribute(&computeMode, hipDeviceAttributeComputeMode, devID));
  CUASSERT(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, devID));
  CUASSERT(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, devID));
  return major * 100 + minor * 10;
}

// ALIGNPTR
int8_t *alignPtr(int8_t *ptr, uintptr_t to) {
  uintptr_t addr = (uintptr_t)ptr;
  if (addr % to) {
    addr += to - addr % to;
  }
  return (int8_t *)addr;
}

// NEXTWORKSPACEPTR
int8_t *nextWorkspacePtr(int8_t *ptr, uintptr_t previousWorkspaceSize) {
  uintptr_t addr = (uintptr_t)ptr;
  addr += previousWorkspaceSize;
  return alignPtr((int8_t *)addr, CUDA_MEM_ALIGN);
}

// CALCULATE TOTAL WORKSPACE SIZE
size_t calculateTotalWorkspaceSize(size_t *workspaces, int count) {
  size_t total = 0;
  for (int i = 0; i < count; i++) {
    total += workspaces[i];
    if (workspaces[i] % CUDA_MEM_ALIGN) {
      total += CUDA_MEM_ALIGN - (workspaces[i] % CUDA_MEM_ALIGN);
    }
  }
  return total;
}

using nvinfer1::DataType;

template <unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta) __global__
    void setUniformOffsets_kernel(const int num_segments, const int offset, int *d_offsets) {
  const int idx = blockIdx.x * nthds_per_cta + threadIdx.x;
  if (idx <= num_segments) d_offsets[idx] = idx * offset;
}

void setUniformOffsets(hipStream_t stream, const int num_segments, const int offset,
                       int *d_offsets) {
  const int BS = 32;
  const int GS = (num_segments + 1 + BS - 1) / BS;
  setUniformOffsets_kernel<BS><<<GS, BS, 0, stream>>>(num_segments, offset, d_offsets);
}

size_t detectionForwardBBoxDataSize(int N, int C1, DataType DT_BBOX) {
  if (DT_BBOX == DataType::kFLOAT) {
    return N * C1 * sizeof(float);
  }

  printf("Only FP32 type bounding boxes are supported.\n");
  return (size_t)-1;
}

size_t detectionForwardBBoxPermuteSize(bool shareLocation, int N, int C1, DataType DT_BBOX) {
  if (DT_BBOX == DataType::kFLOAT) {
    return shareLocation ? 0 : N * C1 * sizeof(float);
  }
  printf("Only FP32 type bounding boxes are supported.\n");
  return (size_t)-1;
}

size_t detectionForwardPreNMSSize(int N, int C2) {
  ASSERT(sizeof(float) == sizeof(int));
  return N * C2 * sizeof(float);
}

size_t detectionForwardPostNMSSize(int N, int numClasses, int topK) {
  ASSERT(sizeof(float) == sizeof(int));
  return N * numClasses * topK * sizeof(float);
}

size_t detectionInferenceWorkspaceSize(bool shareLocation, int N, int C1, int C2, int numClasses,
                                       int numPredsPerClass, int topK, DataType DT_BBOX,
                                       DataType DT_SCORE) {
  size_t wss[7];
  wss[0] = detectionForwardBBoxDataSize(N, C1, DT_BBOX);
  wss[1] = detectionForwardBBoxPermuteSize(shareLocation, N, C1, DT_BBOX);
  wss[2] = detectionForwardPreNMSSize(N, C2);
  wss[3] = detectionForwardPreNMSSize(N, C2);
  wss[4] = detectionForwardPostNMSSize(N, numClasses, topK);
  wss[5] = detectionForwardPostNMSSize(N, numClasses, topK);
  wss[6] = std::max(sortScoresPerClassWorkspaceSize(N, numClasses, numPredsPerClass, DT_SCORE),
                    sortScoresPerImageWorkspaceSize(N, numClasses * topK, DT_SCORE));
  return calculateTotalWorkspaceSize(wss, 7);
}
