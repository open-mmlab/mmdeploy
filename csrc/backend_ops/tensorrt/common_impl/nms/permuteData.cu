#include "hip/hip_runtime.h"
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
// modify from
// https://github.com/NVIDIA/TensorRT/tree/master/plugin/batchedNMSPlugin
#include <vector>

#include "nms/kernel.h"

template <typename Dtype, unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta) __global__
    void permuteData_kernel(const int nthreads, const int num_classes, const int num_data,
                            const int num_dim, bool confSigmoid, const Dtype *data,
                            Dtype *new_data) {
  // data format: [batch_size, num_data, num_classes, num_dim]
  for (int index = blockIdx.x * nthds_per_cta + threadIdx.x; index < nthreads;
       index += nthds_per_cta * gridDim.x) {
    const int i = index % num_dim;
    const int c = (index / num_dim) % num_classes;
    const int d = (index / num_dim / num_classes) % num_data;
    const int n = index / num_dim / num_classes / num_data;
    const int new_index = ((n * num_classes + c) * num_data + d) * num_dim + i;
    float result = data[index];
    if (confSigmoid) result = exp(result) / (1 + exp(result));

    new_data[new_index] = result;
  }
  // new data format: [batch_size, num_classes, num_data, num_dim]
}

template <typename Dtype>
pluginStatus_t permuteData_gpu(hipStream_t stream, const int nthreads, const int num_classes,
                               const int num_data, const int num_dim, bool confSigmoid,
                               const void *data, void *new_data) {
  const int BS = 512;
  const int GS = (nthreads + BS - 1) / BS;
  permuteData_kernel<Dtype, BS><<<GS, BS, 0, stream>>>(nthreads, num_classes, num_data, num_dim,
                                                       confSigmoid, (const Dtype *)data,
                                                       (Dtype *)new_data);
  CSC(hipGetLastError(), STATUS_FAILURE);
  return STATUS_SUCCESS;
}

// permuteData LAUNCH CONFIG
typedef pluginStatus_t (*pdFunc)(hipStream_t, const int, const int, const int, const int, bool,
                                 const void *, void *);

struct pdLaunchConfig {
  DataType t_data;
  pdFunc function;

  pdLaunchConfig(DataType t_data) : t_data(t_data) {}
  pdLaunchConfig(DataType t_data, pdFunc function) : t_data(t_data), function(function) {}
  bool operator==(const pdLaunchConfig &other) { return t_data == other.t_data; }
};

static std::vector<pdLaunchConfig> pdFuncVec;

bool permuteDataInit() {
  pdFuncVec.push_back(pdLaunchConfig(DataType::kFLOAT, permuteData_gpu<float>));
  return true;
}

static bool initialized = permuteDataInit();

pluginStatus_t permuteData(hipStream_t stream, const int nthreads, const int num_classes,
                           const int num_data, const int num_dim, const DataType DT_DATA,
                           bool confSigmoid, const void *data, void *new_data) {
  pdLaunchConfig lc = pdLaunchConfig(DT_DATA);
  for (unsigned i = 0; i < pdFuncVec.size(); ++i) {
    if (lc == pdFuncVec[i]) {
      DEBUG_PRINTF("permuteData kernel %d\n", i);
      return pdFuncVec[i].function(stream, nthreads, num_classes, num_data, num_dim, confSigmoid,
                                   data, new_data);
    }
  }
  return STATUS_BAD_PARAM;
}
